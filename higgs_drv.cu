#include "hip/hip_runtime.h"

#define _CUDA_READY_

#include "common.h"

#include "higgs.cc"
#include "higgscouplings.cc"
#include "higgshelas.cc"
#include "pdf.cc"

extern "C"
{
  __global__ void quadme(
			  afloat_t* result, afloat_t* input, afloat_t* grd, int* idxdim, int ndim, /* phase space data */
			  afloat_t* gpdf_data, int gpdf_nptsx, int gpdf_nptsq, afloat_t gpdf_lxmin, afloat_t gpdf_lxmax, afloat_t gpdf_lqmin, afloat_t gpdf_lqmax, /* gluon PDF data table */
			  afloat_t ca, afloat_t kSM, afloat_t kHWW, afloat_t kAWW, afloat_t lambda /* Higgs characterization model parameters */
			 )
  {
    // __shared__ struct couplings _couplings;
    // if(threadIdx.x==0)
    // {
    //   init(&_couplings, ca, kSM, kHWW, kAWW, lambda);
    // }
    // __syncthreads();

    struct couplings _couplings;
    init(&_couplings, ca, kSM, kHWW, kAWW, lambda);
    
    int idxthr = threadIdx.x + blockIdx.x * blockDim.x;
     
    const uint nkin = 4*nexternal;

    const uint IP0_E = 4*0; const uint IP0_X = 4*0+1; const uint IP0_Y = 4*0+2; const uint IP0_Z = 4*0+3;
    const uint IP1_E = 4*1; const uint IP1_X = 4*1+1; const uint IP1_Y = 4*1+2; const uint IP1_Z = 4*1+3;
    const uint IP2_E = 4*2; const uint IP2_X = 4*2+1; const uint IP2_Y = 4*2+2; const uint IP2_Z = 4*2+3;
    const uint IP3_E = 4*3; const uint IP3_X = 4*3+1; const uint IP3_Y = 4*3+2; const uint IP3_Z = 4*3+3;
    const uint IP4_E = 4*4; const uint IP4_X = 4*4+1; const uint IP4_Y = 4*4+2; const uint IP4_Z = 4*4+3;
    const uint IP5_E = 4*5; const uint IP5_X = 4*5+1; const uint IP5_Y = 4*5+2; const uint IP5_Z = 4*5+3;
    
    afloat_t p[nkin];
    
    if(ndim==0)
    {
#pragma unroll
      for(unsigned int i=0; i<nkin; ++i)
      {
	p[i] = input[nkin*idxthr+i]; // no integration
      }
    }    
    else
    {
#pragma unroll
      for(unsigned int i=0; i<nkin; ++i)
      {
	p[i] = input[i]; /* make copy of p a shared memory object */
      }
      for(unsigned int i=0; i<ndim; ++i)
      {   
	p[idxdim[i]] = grd[ndim*idxthr+i]; // set momenta from integration
      }

      // ensure E,p make sense for final state particles
      
      p[IP5_E] = _SQRT_(_POW_(p[IP5_X],2)+_POW_(p[IP5_Y],2)+_POW_(p[IP5_Z],2));
      p[IP4_E] = _SQRT_(_POW_(p[IP4_X],2)+_POW_(p[IP4_Y],2)+_POW_(p[IP4_Z],2));
      p[IP3_E] = _SQRT_(_POW_(p[IP3_X],2)+_POW_(p[IP3_Y],2)+_POW_(p[IP3_Z],2));
      p[IP2_E] = _SQRT_(_POW_(p[IP2_X],2)+_POW_(p[IP2_Y],2)+_POW_(p[IP2_Z],2));
    }
    
    // solve for the initial state of 2 -> 4 process

    afloat_t px_tot = p[IP2_X]+p[IP3_X]+p[IP4_X]+p[IP5_X];
    afloat_t py_tot = p[IP2_Y]+p[IP3_Y]+p[IP4_Y]+p[IP5_Y];
    afloat_t pz_tot = p[IP2_Z]+p[IP3_Z]+p[IP4_Z]+p[IP5_Z];
    afloat_t E_tot  = p[IP2_E]+p[IP3_E]+p[IP4_E]+p[IP5_E];

    afloat_t msqr_tot = _POW_(E_tot,2) - _POW_(pz_tot,2) - _POW_(py_tot,2) - _POW_(px_tot,2);

    afloat_t tmp = _SQRT_(_POW_(pz_tot,2) + msqr_tot);
    
    afloat_t e0 = (tmp + pz_tot)/2;
    afloat_t e1 = (tmp - pz_tot)/2;

    p[IP0_E] = e0; p[IP0_Z] =  e0; p[IP0_Y] = 0; p[IP0_X] = 0;
    p[IP1_E] = e1; p[IP1_Z] = -e1; p[IP1_Y] = 0; p[IP1_X] = 0;

    if(e0 > EBEAM || e1 > EBEAM)
    {
      result[idxthr] = 0;
      return;
    }         
   
    // afloat_t qsqr = _POW_(p[IP0_E]+p[IP1_E],2) - _POW_(p[IP0_Z]+p[IP1_Z],2);

    afloat_t scale = 1.0; // keep dLIPS^{-1} from blowing up ... FP precision issues

    afloat_t dLIPS = 1.0;
#pragma unroll
    for(unsigned int ip=2; ip<nexternal; ++ip)
    {
      dLIPS *= ( (2*p[ip*4]) / scale );
    }
    
    // calculate the matrix element and PDF

    result[idxthr] = ( (me(p, 0x0, 0, &_couplings)) *
		       (pdf(p[IP0_E]/EBEAM, _SQRT_(msqr_tot), gpdf_data, gpdf_nptsx, gpdf_nptsq, gpdf_lxmin, gpdf_lxmax, gpdf_lqmin, gpdf_lqmax)) *
		       (pdf(p[IP1_E]/EBEAM, _SQRT_(msqr_tot), gpdf_data, gpdf_nptsx, gpdf_nptsq, gpdf_lxmin, gpdf_lxmax, gpdf_lqmin, gpdf_lqmax)) / (dLIPS) );
  }
}
